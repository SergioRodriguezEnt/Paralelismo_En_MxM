#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <papi.h>
#include <hip/hip_runtime.h>
#define hl_region_begin(A) retval = PAPI_hl_region_begin(A); \
        if ( retval != PAPI_OK ) handle_error(retval);
#define hl_region_end(A) retval = PAPI_hl_region_end(A); \
        if ( retval != PAPI_OK ) handle_error(retval);
//nvcc MxM.cu -I/${PAPI_DIR}/include -L/${PAPI_DIR}/lib -o MxM -lpapi

//
#define BLOCK_SIZE 5
//Matrices nxn
#define n 250
float A[n*n];
float B[n*n];
float C[n*n]; // C = A*B
//Iteraciones a repetir el programa
int Niter = 10000;

void Inicializar_Matrices(float* x, float* y, float* z, int size) {
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            float num = (float) (i+j);
            x[i*size+j] = num;
            y[i*size+j] = num;
            z[i*size+j] = 0.;
        }
    }
}

void Imprimir_Matriz(float* m, int size) {
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            printf("%f\t", m[i*size+j]);
        }
        printf("\n");
    }
    printf("\n");
}

void Imprimir_Inicio(float* x, float* y, int size) {
    printf("#################################\n");
    printf("# PROGRAMA MATRIZ x MATRIZ BASE #\n");
    printf("#################################\n\n");
    printf("LA MATRIZ A ES:\n");
    Imprimir_Matriz(x, size);
    printf("LA MATRIZ B ES:\n");
    Imprimir_Matriz(x, size);
    printf("\nCOMIENZA LA EJECUCION\n");
}

void Imprimir_Resultados(float* z, int size, int iters) {
    printf("EJECUCION TERMINADA\n");
    printf("LA MATRIZ RESULTANTE C ES:\n");
    Imprimir_Matriz(z, size);
    printf("SE HAN REALIZADO %d ITERACIONES.\n", iters);
}

__global__ void Matriz_Matriz_kernel(float* x, float* y, float* z, int size) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    float num = 0.;
    if (i < size && j < size) {
        for (int k = 0; k < size; k++) {
            num += x[i*size+k] * y [k*size+j];
        }
        z[i*size+j] = num;
    }
}

void CUDA_CHECK(hipError_t err) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d \n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(1);
    }
}

void handle_error(int retval) {
    printf("PAPI error %d: %s\n", retval, PAPI_strerror(retval));
    exit(1);
}

int main() {
    int retval;
    hl_region_begin("program");
    
    hl_region_begin("initialization");
    Inicializar_Matrices(A, B, C, n);
    float* cudaA;
    float* cudaB;
    float* cudaC;
    int arr_size = n*n*sizeof(float);
    hipError_t err = hipMalloc((void**)&cudaA, arr_size);
    CUDA_CHECK(err);
    hipMalloc((void**)&cudaB, arr_size);
    CUDA_CHECK(err);
    hipMalloc((void**)&cudaC, arr_size);
    CUDA_CHECK(err);
    hipMemcpy(cudaA, A, arr_size, hipMemcpyHostToDevice);
    CUDA_CHECK(err);
    hipMemcpy(cudaB, B, arr_size, hipMemcpyHostToDevice);
    CUDA_CHECK(err);
    hipMemcpy(cudaC, C, arr_size, hipMemcpyHostToDevice);
    CUDA_CHECK(err);
    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 dim_grid(ceil(n/BLOCK_SIZE), ceil(n/BLOCK_SIZE), 1);
    Imprimir_Inicio(A, B, n);
    hl_region_end("initialization");
    
    hl_region_begin("computation");
    for (int i = 0; i < Niter; i++) {
        Matriz_Matriz_kernel<<<dim_grid, dim_block>>> (cudaA, cudaB, cudaC, n);
        hipDeviceSynchronize();
    }
    hl_region_end("computation");
    
    hl_region_begin("end");
    hipMemcpy(C, cudaC, arr_size, hipMemcpyDeviceToHost);
    CUDA_CHECK(err);
    hipFree(cudaA);
    hipFree(cudaB);
    hipFree(cudaC);
    Imprimir_Resultados(C, n, Niter);
    hl_region_end("end");
    
    hl_region_end("program");
}